
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>

#ifndef block_size_x
#define block_size_x 256
#endif

#ifndef window_width
#define window_width 1500
#endif

#ifndef write_rows
#define write_rows 1
#endif 

#ifndef shared_memory_size
#define shared_memory_size 12*block_size_x
#endif

/*
 * This kernel creates a sparse representation of the densely stored correlations table.
 *
 * In addition to the correlations table, this kernel needs a precomputed prefix_sums array. This
 * array contains the inclusive prefix sums of the degrees of the nodes in the correlations table.
 * In other words, it is an array with one element per hit, containing the sum over the total
 * number of hits correlated with the hits up to and including that hit in the correlations table.
 *
 * Output arguments are row_idx and col_idx, which contain the (hit id, hit id) pairs that describe
 * the correlated hits.
 *
 */
__global__ void dense2sparse_kernel(int *row_idx, int *__restrict__ col_idx, int *__restrict__ prefix_sums, uint8_t * correlations, int n) {
    int i = blockIdx.x * block_size_x + threadIdx.x;

    #if use_shared == 1
    __shared__ int sh_col_idx[shared_memory_size];
    int block_start = 0;
    if (blockIdx.x > 0) {
        block_start = prefix_sums[blockIdx.x * block_size_x - 1];
    }
    #endif

    if (i<n) {
        //get the offset to where output should be written
        int offset = 0;
        if (i>0) {
            offset = prefix_sums[i-1];
        }

        //see how much work there is on this row
        //int end = prefix_sums[i];

        //collect the edges to nodes with lower id
        if (i<window_width) {
            for (int j=i-1; j>=0; j--) {
                int col = i-j-1;
                uint64_t pos = (j * (uint64_t)n) + (uint64_t) (col);
                if (correlations[pos] == 1) {
                    #if write_rows
                    row_idx[offset] = i;
                    #endif

                    #if use_shared == 1
                    sh_col_idx[offset - block_start] = col;
                    #else
                    col_idx[offset] = col;
                    #endif
                    offset += 1;
                }
            }
        } else {
            #if f_unroll == 2
            #pragma unroll 2
            #elif f_unroll == 3
            #pragma unroll 3
            #elif f_unroll == 4
            #pragma unroll 4
            #elif f_unroll == 5
            #pragma unroll 5
            #elif f_unroll == 6
            #pragma unroll 6
            #endif
            for (int j=window_width-1; j>=0; j--) {
                int col = i-j-1;
                uint64_t pos = (j * (uint64_t)n) + (uint64_t) (col);
                if (correlations[pos] == 1) {
                    #if write_rows
                    row_idx[offset] = i;
                    #endif

                    #if use_shared == 1
                    sh_col_idx[offset - block_start] = col;
                    #else
                    col_idx[offset] = col;
                    #endif
                    offset += 1;
                }
            }
        }

        //collect the edges to nodes with higher id
        #if f_unroll == 2
        #pragma unroll 2
        #elif f_unroll == 3
        #pragma unroll 3
        #elif f_unroll == 4
        #pragma unroll 4
        #elif f_unroll == 5
        #pragma unroll 5
        #elif f_unroll == 6
        #pragma unroll 6
        #endif
        for (int j=0; j<window_width; j++) {
            uint64_t pos = (j * (uint64_t)n) + (uint64_t)i;
            if (correlations[pos] == 1) {
                #if write_rows
                row_idx[offset] = i;
                #endif

                #if use_shared == 1
                sh_col_idx[offset - block_start] = i+j+1;
                #else
                col_idx[offset] = i+j+1;
                #endif
                offset += 1;
            }
        }

    }



    //collaboratively write back the output collected in shared memory to global memory
    #if use_shared == 1
    int block_stop;
    int last_i = blockIdx.x * block_size_x + block_size_x-1;
    if (last_i < n) {
        block_stop = prefix_sums[last_i];
    } else {
        block_stop = prefix_sums[n-1];
    }
    __syncthreads(); //ensure all threads are done writing shared memory
    for (int k=block_start+threadIdx.x; k<block_stop; k+=block_size_x) {
        col_idx[k] = sh_col_idx[k-block_start];
    }

    #endif


}

