
#include <hip/hip_runtime.h>
#include <inttypes.h>

#ifndef block_size_x
#define block_size_x 256
#endif

#ifndef window_width
#define window_width 1500
#endif

#ifndef write_rows
#define write_rows 1
#endif 

/*
 * This kernel creates a sparse representation of the densely stored correlations table.
 *
 * In addition to the correlations table, this kernel needs a precomputed prefix_sums array. This
 * array contains the inclusive prefix sums of the degrees of the nodes in the correlations table.
 * In other words, it is an array with one element per hit, containing the sum over the total
 * number of hits correlated with the hits up to and including that hit in the correlations table.
 *
 * Output arguments are row_idx and col_idx, which contain the (hit id, hit id) pairs that describe
 * the correlated hits.
 *
 */
__global__ void dense2sparse_kernel(int *row_idx, int *__restrict__ col_idx, int *__restrict__ prefix_sums, uint8_t * correlations, int n) {
    int i = blockIdx.x * block_size_x + threadIdx.x;

    if (i<n) {
        //get the offset to where output should be written
        int offset = 0;
        if (i>0) {
            offset = prefix_sums[i-1];
        }

        //see how much work there is on this row
        //int end = prefix_sums[i];

        //collect the edges to nodes with lower id
        if (i<window_width) {
            for (int j=i-1; j>=0; j--) {
                int col = i-j-1;
                uint64_t pos = (j * (uint64_t)n) + (uint64_t) (col);
                if (correlations[pos] == 1) {
                    #if write_rows
                    row_idx[offset] = i;
                    #endif
                    col_idx[offset] = col;
                    offset += 1;
                }
            }
        } else {
            #if f_unroll == 2
            #pragma unroll 2
            #elif f_unroll == 3
            #pragma unroll 3
            #elif f_unroll == 4
            #pragma unroll 4
            #elif f_unroll == 5
            #pragma unroll 5
            #elif f_unroll == 6
            #pragma unroll 6
            #endif
            for (int j=window_width-1; j>=0; j--) {
                int col = i-j-1;
                uint64_t pos = (j * (uint64_t)n) + (uint64_t) (col);
                if (correlations[pos] == 1) {
                    #if write_rows
                    row_idx[offset] = i;
                    #endif
                    col_idx[offset] = col;
                    offset += 1;
                }
            }
        }


        //collect the edges to nodes with higher id
        #if f_unroll == 2
        #pragma unroll 2
        #elif f_unroll == 3
        #pragma unroll 3
        #elif f_unroll == 4
        #pragma unroll 4
        #elif f_unroll == 5
        #pragma unroll 5
        #elif f_unroll == 6
        #pragma unroll 6
        #endif
        for (int j=0; j<window_width; j++) {
            uint64_t pos = (j * (uint64_t)n) + (uint64_t)i;
            if (correlations[pos] == 1) {
                #if write_rows
                row_idx[offset] = i;
                #endif
                col_idx[offset] = i+j+1;
                offset += 1;
            }
        }


    }
}

